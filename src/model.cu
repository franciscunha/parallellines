#include "hip/hip_runtime.h"
// This file is forked from an initial version by Dmitry V. Sokolov (https://github.com/ssloy) that was
// provided as a starting point for the tinyrenderer course (https://github.com/ssloy/tinyrenderer)

#include <iostream>
#include <string>
#include <fstream>
#include <sstream>
#include <vector>
#include <algorithm>
#include "../include/model.cuh"

Model::Model(const char *filename)
{
    std::ifstream in;
    in.open(filename, std::ifstream::in);
    if (in.fail())
    {
        std::cerr << "failed to open " << filename << std::endl;
        return;
    }

    std::vector<std::vector<int>> faces;
    std::vector<std::vector<int>> faces_uvs;
    std::vector<std::vector<int>> faces_normals;
    std::vector<Vec3f> verts;
    std::vector<Vec3f> uvs;
    std::vector<Vec3f> normals;

    std::string line;
    while (!in.eof())
    {
        std::getline(in, line);
        std::istringstream iss(line.c_str());

        char trash;
        if (!line.compare(0, 2, "v "))
        {
            iss >> trash;
            Vec3f v;
            for (int i = 0; i < 3; i++)
                iss >> v.raw[i];
            verts.push_back(v);
        }
        else if (!line.compare(0, 3, "vt "))
        {
            iss >> trash >> trash;
            Vec3f vt;
            vt.raw[2] = 0;
            for (int i = 0; i < 2; i++)
                iss >> vt.raw[i];
            uvs.push_back(vt);
        }
        else if (!line.compare(0, 3, "vn "))
        {
            iss >> trash >> trash;
            Vec3f vn;
            for (int i = 0; i < 3; i++)
                iss >> vn.raw[i];
            normals.push_back(vn);
        }
        else if (!line.compare(0, 2, "f "))
        {
            std::vector<int> v_indexes;
            std::vector<int> t_indexes;
            std::vector<int> n_indexes;
            int v_idx, t_idx, n_idx;
            iss >> trash;
            while (iss >> v_idx >> trash >> t_idx >> trash >> n_idx)
            {
                // in wavefront obj all indices start at 1, not zero
                v_idx--;
                t_idx--;
                n_idx--;

                v_indexes.push_back(v_idx);
                t_indexes.push_back(t_idx);
                n_indexes.push_back(n_idx);
            }
            faces.push_back(v_indexes);
            faces_uvs.push_back(t_indexes);
            faces_normals.push_back(n_indexes);
        }
    }

    n_verts_ = verts.size();
    n_uvs_ = uvs.size();
    n_normals_ = normals.size();

    vectors_ = new Vec3f[n_verts_ + n_uvs_ + n_normals_];
    std::copy(verts.begin(), verts.end(), vectors_);
    std::copy(uvs.begin(), uvs.end(), vectors_ + n_verts_);
    std::copy(normals.begin(), normals.end(), vectors_ + n_verts_ + n_uvs_);

    n_faces_ = faces.size();

    indexes_ = new int[n_faces_ * 9];
    for (int i = 0; i < n_faces_; i++)
    {
        for (int j = 0; j < 3; j++)
        {
            indexes_[(0 * 3 * n_faces_) + (i * 3 + j)] = faces[i][j];
            indexes_[(1 * 3 * n_faces_) + (i * 3 + j)] = faces_uvs[i][j];
            indexes_[(2 * 3 * n_faces_) + (i * 3 + j)] = faces_normals[i][j];
        }
    }

    diffuse_ = new TGAImage();
    normal_map_ = new TGAImage();
    specular_ = new TGAImage();
}

Model::~Model()
{
    delete[] indexes_;
    delete[] vectors_;

    delete diffuse_;
    delete normal_map_;
    delete specular_;
}

Model *Model::cudaDeepCopyToDevice()
{
    // make device copies of the data in this

    int *d_indexes;
    Vec3f *d_vectors;
    TGAImage *d_diffuse;
    TGAImage *d_specular;
    TGAImage *d_normal_map;

    int size_indexes = (n_faces_ * 9) * sizeof(int);
    int size_vectors = (n_verts_ + n_uvs_ + n_normals_) * sizeof(Vec3f);

    hipMalloc(&d_indexes, size_indexes);
    hipMalloc(&d_vectors, size_vectors);

    hipMemcpy(d_indexes, indexes_, size_indexes, hipMemcpyHostToDevice);
    hipMemcpy(d_vectors, vectors_, size_vectors, hipMemcpyHostToDevice);

    // textures need deep copies
    if (diffuse_)
        d_diffuse = diffuse_->cudaDeepCopyToDevice();
    if (specular_)
        d_specular = specular_->cudaDeepCopyToDevice();
    if (normal_map_)
        d_normal_map = normal_map_->cudaDeepCopyToDevice();

    // temporarily make this point to the device data s.t. device model points to them

    int *indexes = indexes_;
    Vec3f *vectors = vectors_;
    TGAImage *diffuse = diffuse_;
    TGAImage *specular = specular_;
    TGAImage *normal_map = normal_map_;

    indexes_ = d_indexes;
    vectors_ = d_vectors;
    diffuse_ = d_diffuse;
    specular_ = d_specular;
    normal_map_ = d_normal_map;

    // copy this to device

    Model *d_model;
    hipMalloc(&d_model, sizeof(Model));
    hipMemcpy(d_model, this, sizeof(Model), hipMemcpyHostToDevice);

    // reset the host model pointers to the host arrays

    indexes_ = indexes;
    vectors_ = vectors;
    diffuse_ = diffuse;
    specular_ = specular;
    normal_map_ = normal_map;

    // return pointer to device model
    return d_model;
}

void Model::cudaDeepFree(Model *device_ptr)
{
    // copy device data to host so we can access the inner addresses
    Model *m = (Model *)malloc(sizeof(Model));
    hipMemcpy(m, device_ptr, sizeof(Model), hipMemcpyDeviceToHost);

    // free everything
    hipFree(m->indexes_);
    hipFree(m->vectors_);
    free(m);
    hipFree(device_ptr);

    if (m->diffuse_)
        TGAImage::cudaDeepFree(m->diffuse_);
    if (m->normal_map_)
        TGAImage::cudaDeepFree(m->normal_map_);
    if (m->specular_)
        TGAImage::cudaDeepFree(m->specular_);
}

__host__ __device__ TGAImage *Model::texture_of_type(TextureType type)
{
    switch (type)
    {
    case TextureType::DIFFUSE:
        return diffuse_;
    case TextureType::NORMAL_MAP:
        return normal_map_;
    case TextureType::SPECULAR:
        return specular_;
    }
    return diffuse_;
}

void Model::load_texture(const char *filename, TextureType type)
{
    TGAImage *texture = texture_of_type(type);

    texture->read_tga_file(filename);
    texture->flip_vertically(); // so the origin is left bottom corner
}

__host__ __device__ TGAColor Model::sample_texture(Vec2f uv, TextureType type)
{
    TGAImage *texture = texture_of_type(type);
    if (texture == nullptr)
    {
        return TGAColor(0, 0, 0, 0);
    }

    return texture->get(
        std::round(uv.x * (float)texture->get_width()),
        std::round(uv.y * (float)texture->get_height()));
}
